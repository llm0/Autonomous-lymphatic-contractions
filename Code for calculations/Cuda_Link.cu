#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "lymph_h.h"
#include "math.h"
//#include "StdAfx.h"
#include "Cuda_Link.h"
__device__ Cuda_Link& Cuda_Link:: operator =(const Cuda_Link &v)
{
	m_iSegf = v.m_iSegf;
	m_iSegs = v.m_iSegs;
	m_iType = v.m_iType;
	m_iSolx = v.m_iSolx;
	m_iSoly = v.m_iSoly;
	m_iFlux = v.m_iFlux;
	m_iFluy = v.m_iFluy;
	m_iDir = v.m_iDir;
	m_bf = v.m_bf;
	m_bs = v.m_bs;
	m_dDlf = v.m_dDlf;
	m_dDls = v.m_dDls;
	m_bdb = v.m_bdb;
	return *this;
}