#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "lymph_h.h"
#include "math.h"
//#include "StdAfx.h"
#include "Cuda_Segment.h"

__device__ void Cuda_Segment::init(Cuda_Segment* mem, int* iSeg, parameter * p)//���ķ���Ϊ����ָ����ڣ����⿴Ϊ����
{
	if (*iSeg != 0)
	{
		if (p->iMyID == 0 && *iSeg >= p->iBegin_Mem)
		{
			this->m_prev = NULL;
		}
		else
		{
			this->m_prev = &mem[*iSeg - 1];
		}
	}
	else
	{
		this->m_prev = NULL;
	}
	
	if (*iSeg != SEGNUM_E - 1 )
	{
		if (p->iMyID == p->iNumProcs - 1 && *iSeg == p->iEnd_Mem-1)
		{
			this->m_next = NULL;
		}
		else
		{
			this->m_next = &mem[*iSeg + 1];
		}
	}
	else
	{
		this->m_next = NULL;
	}
}
