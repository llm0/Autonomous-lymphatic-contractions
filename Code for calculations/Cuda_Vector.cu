#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "lymph_h.h"
#include "math.h"
//#include "StdAfx.h"
#include "Cuda_Vector.h"

__device__  Cuda_Vector::Cuda_Vector(double x,double y)
{
	m_dx = x;
	m_dy = y;
}
__device__ Cuda_Vector& Cuda_Vector:: operator =(const Cuda_Vector &v)
{
	m_dx = v.m_dx;
	m_dy = v.m_dy;
	return *this;
}
__device__ Cuda_Vector& Cuda_Vector:: operator =(const Vector &v)
{
	m_dx = v.m_dx;
	m_dy = v.m_dy;
	return *this;
}

__device__ Cuda_Vector Cuda_Vector:: operator-(Cuda_Vector v)
{
	Cuda_Vector R;
	R.m_dx = m_dx - v.m_dx;
	R.m_dy = m_dy - v.m_dy;
	return R;
}

__device__ double Cuda_Vector::operator^(const Cuda_Vector &v)
{
	double dR;
	dR = m_dx*v.m_dy - m_dy*v.m_dx;
	return dR;
}

__device__ double Cuda_Vector::operator*(const Cuda_Vector &v)
{
	double dR;
	dR = m_dx*v.m_dx + m_dy*v.m_dy;
	return dR;
}