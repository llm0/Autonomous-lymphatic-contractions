#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "lymph_h.h"
#include "kernel.h"
#include "math.h"
//#include "StdAfx.h"
#include "Cuda_Link.cu" 
#include "Cuda_Segment.cu" 
#include "Cuda_Vector.cu" 


#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <fstream>
using namespace std;

double dTest;
int iMemSize = sizeof(lattice_str)*WIDTH_E*HEIGHT;


//parameter* dev_p;//���е�ָ�붼����host�ϣ�cuda���ڴ��� GPU��
parameter *dev_p;
lattice_str(*dev_a)[HEIGHT] = NULL;
lattice_str(*dev_a_bak)[HEIGHT] = NULL;
double(*dev_NOBuf)[HEIGHT] = NULL;
Cuda_Segment* dev_sm1 = NULL;//up
Cuda_Segment* dev_sm2 = NULL;//down
Cuda_Link *dev_link = NULL;
int* dev_linknum;
lattice_link(*dev_laln)[HEIGHT] = NULL;
int *dev_ixn;
int *dev_iyn;
Valve_str * dev_valve;
double *dev_test;





extern "C" int InitCudaParameter(int gpuid, parameter* InitData, lattice_str la[WIDTH_E][HEIGHT], Cuda_Segment memup[SEGNUM_E],
	Cuda_Segment memdown[SEGNUM_E], double pNO[WIDTH_E][HEIGHT], Cuda_Link* plink, int iMyID)
{
	hipError_t cudaStatus;
	int devID, iCount = 8;
	hipDeviceProp_t deviceProp;
	cudaStatus = hipGetDeviceCount(&iCount);
	printf("CudaNumber is %d\n", iCount);
	if (iCount > 0)
	{
		devID = (iMyID % 8) % iCount;
	}
	else
	{
		return 0;
	}
	hipGetDeviceProperties(&deviceProp, devID);
	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}
	cudaStatus = hipSetDevice(devID);
	printf("Use GPU %d\n", devID);
	cudaStatus = hipMalloc((void**)&dev_valve, sizeof(Valve_str));
	cudaStatus = hipMalloc((void**)&dev_p, sizeof(parameter));
	cudaStatus = hipMemcpy((void**)dev_p, InitData, sizeof(parameter), hipMemcpyHostToDevice);
	//cudaStatus = hipMalloc((void**)&dev_a, iMemSize);
	cudaStatus = hipMalloc((void**)&dev_a_bak, iMemSize);
	cudaStatus = hipMalloc((void**)&dev_sm1, SEGNUM_E *  sizeof(Cuda_Segment));//���ڴ�
	cudaStatus = hipMalloc((void**)&dev_sm2, SEGNUM_E *  sizeof(Cuda_Segment));//���ڴ�
	//cudaStatus = hipMalloc((void**)& dev_NOBuf, WIDTH * HEIGHT* sizeof(double));//���ڴ�
	//cudaStatus = hipMalloc((void**)& dev_link, LINKNUM* sizeof(Cuda_Link));//���ڴ�
	cudaStatus = hipMalloc((void**)& dev_linknum, sizeof(int));//���ڴ�
	cudaStatus = hipMalloc((void**)& dev_laln, WIDTH_E * HEIGHT*sizeof(lattice_link));//���ڴ�
	cudaStatus = hipMalloc((void**)& dev_ixn, sizeof(int));
	cudaStatus = hipMalloc((void**)& dev_iyn, sizeof(int));
	cudaStatus = hipMalloc((void**)& dev_test, sizeof(double));
	cudaStatus = hipHostRegister(la, iMemSize,
		hipHostRegisterMapped); if (cudaStatus != hipSuccess)	{ return 0; }
	//cudaStatus = hipHostRegister(memup, SEGNUM *  sizeof(Cuda_Segment),
	//	hipHostRegisterMapped); if (cudaStatus != hipSuccess)	{ return 0; }
	//cudaStatus = hipHostRegister(memdown, SEGNUM *  sizeof(Cuda_Segment),
	//	hipHostRegisterMapped); if (cudaStatus != hipSuccess)	{ return 0; }
	cudaStatus = hipHostRegister(pNO, WIDTH_E*HEIGHT*sizeof(double),
		hipHostRegisterMapped); if (cudaStatus != hipSuccess)	{ return 0; }
	cudaStatus = hipHostRegister(plink, LINKNUM* sizeof(Cuda_Link),
		hipHostRegisterMapped); if (cudaStatus != hipSuccess)	{ return 0; }
	cudaStatus = hipHostGetDevicePointer((void**)&dev_a, la, 0); if (cudaStatus != hipSuccess)	{ return 0; }
	//cudaStatus = hipHostGetDevicePointer((void**)&dev_sm1, memup, 0); if (cudaStatus != hipSuccess)	{ return 0; }
	//cudaStatus = hipHostGetDevicePointer((void**)&dev_sm2, memdown, 0); if (cudaStatus != hipSuccess)	{ return 0; }
	cudaStatus = hipHostGetDevicePointer((void**)& dev_NOBuf, pNO, 0); if (cudaStatus != hipSuccess)	{ return 0; }
	cudaStatus = hipHostGetDevicePointer((void**)& dev_link, plink, 0); if (cudaStatus != hipSuccess)	{ return 0; }
	cudaStatus = hipGetLastError(); if (cudaStatus != hipSuccess)	{ return 0; }
	if (cudaStatus != hipSuccess)
	{
		return 0;
	}
	else
	{
		return 1;
	}
}

//�ӿں���
extern "C" int CudaCollide()
{
	hipError_t cudaStatus = addWithCuda(1);//Collide ID:1
	if (cudaStatus != hipSuccess)
	{
		return 0;
	}
	return 1;
}

extern "C" int CudaCalculate()
{
	hipError_t cudaStatus = addWithCuda(2);//Calculate ID:2
	if (cudaStatus != hipSuccess)
	{
		return 0;
	}
	return 1;
}

extern "C" int CudaStream()
{
	hipError_t cudaStatus = addWithCuda(3);//Stream ID:3
	if (cudaStatus != hipSuccess)
	{
		return 0;
	}
	return 1;
}

/*extern "C" int CudaStress(Cuda_Segment memup[SEGNUM], Cuda_Segment memdown[SEGNUM])
{
hipError_t cudaStatus = addWithCuda(4, memup, memdown);//Calculate Stress ID:4
if (cudaStatus != hipSuccess)
{
return 0;
}
return 1;
}*/


extern "C" int CudaReactionNO(Valve_str * valve)
{
	hipError_t cudaStatus;
	dim3 Grid(WIDTH, 1, 1);//The Grid includes 1 Block 
	dim3 Threads(HEIGHT, 1, 1);//1 Block includes W*H Threads
	cudaStatus = hipMemcpy(dev_valve, valve, sizeof(Valve_str), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		return 0;
	}
	addKernelReactionNO << < Grid, Threads >> >(dev_a, dev_sm1, dev_sm2, dev_valve, dev_NOBuf, dev_p);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus == hipSuccess)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

extern "C" int CudaDiffusionNO()
{
	hipError_t cudaStatus = addWithCuda(5);//Diffusion NO ID:5
	if (cudaStatus != hipSuccess)
	{
		return 0;
	}
	return 1;
}

extern "C" int CudaFindLink(int * linknum,
	Cuda_Segment memup[SEGNUM_E], Cuda_Segment memdown[SEGNUM_E], parameter* p)
{
	int i;
	int m_ixn, m_ixm, m_iyn, m_iym;
	dim3 Grid(WIDTH, 1, 1);//The Grid includes 1 Block 
	dim3 Threads(HEIGHT, 1, 1);//1 Block includes W*H Threads
	dim3 gg(2, 1, 1), th(SEGNUM_E, 1, 1);
	hipError_t cudaStatus;
	hipDeviceSynchronize();
	if (p->iMyID == 0)
	{
		m_ixn = int(memup[LEFTVALVE].m_pst.m_dx); m_ixm = int(memup[LEFTVALVE].m_pst.m_dx); m_iyn = int(memdown[LEFTVALVE].m_pst.m_dy); m_iym = int(memup[LEFTVALVE].m_pst.m_dy);
	}
	else
	{
		m_ixn = int(memup[0].m_pst.m_dx); m_ixm = int(memup[0].m_pst.m_dx); m_iyn = int(memdown[0].m_pst.m_dy); m_iym = int(memup[0].m_pst.m_dy);
	}
	for (i = p->iBegin_Mem; i<p->iEnd_Mem; i++)
	{
		if (m_ixn>memup[i].m_pst.m_dx)
		{
			m_ixn = int(memup[i].m_pst.m_dx);
		}
		if (m_ixm<memup[i].m_pst.m_dx)
		{
			m_ixm = int(memup[i].m_pst.m_dx + 0.5);
		}
		if (m_iyn>memdown[i].m_pst.m_dy)
		{
			m_iyn = int(memdown[i].m_pst.m_dy);
		}
		if (m_iym < memup[i].m_pst.m_dy)
		{
			m_iym = int(memup[i].m_pst.m_dy + 0.5);
		}
	}
	m_ixn = (m_ixn <= p->iBegin_Row) ? p->iBegin_Row : m_ixn--; m_ixm = (m_ixm >= p->iEnd_Row - 1) ? p->iEnd_Row - 1 : m_ixm++;
	m_iyn = (m_iyn <= 0) ? 0 : m_iyn--; m_iym = (m_iym >= HEIGHT - 1) ? HEIGHT - 1 : m_iym++;
	cudaStatus = hipMemcpy(dev_ixn, &m_ixn, sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_iyn, &m_iyn, sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_sm1, memup, SEGNUM_E *  sizeof(Cuda_Segment), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_sm2, memdown, SEGNUM_E *  sizeof(Cuda_Segment), hipMemcpyHostToDevice);
	addKernelInitMem << < gg, th >> >(dev_sm1, dev_sm2, dev_p, dev_test);
	hipDeviceSynchronize();
	Grid.x = m_ixm - m_ixn + 1;
	Threads.x = m_iym - m_iyn + 1;
	i = 0;
	cudaStatus = hipMemcpy(dev_linknum, &i, sizeof(int), hipMemcpyHostToDevice);
	addKernelFindLink << < Grid, Threads >> >(dev_a, dev_sm1, dev_sm2,
		dev_laln, dev_ixn, dev_iyn, dev_linknum, dev_p, dev_test);
	hipDeviceSynchronize();
	addKernelCopyLink << < Grid, Threads >> >(dev_laln, dev_ixn, dev_iyn, dev_link, dev_p, dev_test);
	hipDeviceSynchronize();
	cudaStatus = hipMemcpy(linknum, dev_linknum, sizeof(int), hipMemcpyDeviceToHost);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		return 0;
	}
	return 1;
}

//Cuda ���̵���
hipError_t addWithCuda(int iAction)
{
	dim3 Grid(WIDTH, 1, 1);//The Grid includes 1 Block 
	dim3 Threads(HEIGHT, 1, 1);//1 Block includes W*H Threads
	hipError_t cudaStatus;
	//cudaStatus = hipMemcpy(dev_a, la, iMemSize, hipMemcpyHostToDevice); if (cudaStatus != hipSuccess) goto Error; //����ȥ
	switch (iAction)
	{
	case 1://ID:1, Collide
		addKernelCollide << < Grid, Threads >> >(dev_a, dev_p);
		cudaStatus = hipDeviceSynchronize();
		break;
	case 2://ID:2, Calculate
		addKernelCalculate << < Grid, Threads >> >(dev_a, dev_p);
		cudaStatus = hipDeviceSynchronize();
		break;
	case 3://ID:3, Stream
		cudaStatus = hipMemcpy(dev_a_bak, dev_a, iMemSize, hipMemcpyDeviceToDevice);//???
		addKernelStream << < Grid, Threads >> >(dev_a, dev_a_bak, dev_p);
		cudaStatus = hipDeviceSynchronize();
		break;
	case 4://ID:4, Calculate stress
		/*cudaStatus = hipMemcpy(dev_sm1, memup, SEGNUM *  sizeof(Cuda_Segment), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(dev_sm2, memdown, SEGNUM *  sizeof(Cuda_Segment), hipMemcpyHostToDevice);
		addKernelStress << < Grid, Threads >> >(dev_a, dev_sm1, dev_sm2, dev_p);
		cudaStatus = hipDeviceSynchronize();*/
		break;
	case 5://ID:5, Diffusion NO
		//cudaStatus = hipMemcpy(dev_NOBuf, pNO, WIDTH * HEIGHT* sizeof(double), hipMemcpyHostToDevice);
		addKernelDeffusionNO << < Grid, Threads >> >(dev_a, dev_NOBuf, dev_p);
		hipDeviceSynchronize();//�˴�����ͬ����Ȼ����ܼ�����NO
		addKernelAddupNO << < Grid, Threads >> >(dev_a, dev_NOBuf, dev_p);
		cudaStatus = hipDeviceSynchronize();
		break;
	default:;
	}
	cudaStatus = hipDeviceSynchronize();// if (cudaStatus != hipSuccess) goto Error;
	//cudaStatus = hipMemcpy(la, dev_a, iMemSize, hipMemcpyDeviceToHost); //if (cudaStatus != hipSuccess) goto Error;
	if (iAction == 5)
	{
		//hipMemcpy(pNO, dev_NOBuf, WIDTH * HEIGHT* sizeof(double), hipMemcpyDeviceToHost);
	}
	return cudaStatus;
}

__device__ void feq(double* dR, int Order, double Den, double Velx, double Vely, parameter* p)
{
	double dfeq;
	double dDotMet;
	dDotMet = p->iPrjx[Order] * Velx + p->iPrjy[Order] * Vely;
	dfeq = Den*p->dCoe[Order] * (1 + 3 * dDotMet + 4.5*dDotMet*dDotMet -
		1.5*(Velx*Velx + Vely*Vely));
	*dR = dfeq;
}

__global__ void addKernelCollide(lattice_str  la[WIDTH_E][HEIGHT], parameter* p)
{
	int i, j, k;
	double dfeq;
	i = blockIdx.x + p->iBegin_Row;
	j = threadIdx.x;
	if (i < p->iEnd_Row&&j < HEIGHT)
	{
		if (la[i][j].m_iSol < 0 && la[i][j].m_ipr != 2)
		{
			for (k = 0; k < 9; k++)
			{
				feq(&dfeq, k, la[i][j].m_dDen, la[i][j].m_v.m_dx, la[i][j].m_v.m_dy, p);
				la[i][j].m_df[k] = la[i][j].m_df[k] - (la[i][j].m_df[k] - dfeq) / p->Tau
					+ p->dGravity*cos(p->dGravityAngle)*la[i][j].m_dDen* p->iPrjx[k] / 6.0
					+ p->dGravity*sin(p->dGravityAngle)*la[i][j].m_dDen* p->iPrjy[k] / 6.0;
			}
		}
	}
}

__global__ void addKernelCalculate(lattice_str  la[WIDTH_E][HEIGHT], parameter* p)//�Ѿ��üĴ����������ȫ�ֱ���
{
	int i, j, k;
	double dfeq;
	i = blockIdx.x + p->iBegin_Row;
	j = threadIdx.x;
	la[i][j].m_bReaction = 0;//0
	double dDen = 0, dvx = 0, dvy = 0, df[9];//����ֲ�������Cuda�����Ĵ�������
	if (la[i][j].m_iSol < 0 && la[i][j].m_ipr != 2)
	{
		for (k = 0; k < 9; k++)
		{
			df[k] = la[i][j].m_df[k];
		}
		for (k = 0; k<9; k++)
		{
			dDen += df[k];
			dvx += df[k] * p->iPrjx[k];
			dvy += df[k] * p->iPrjy[k];
		}
		if (dDen >10)
		{
			printf("Too large Density Proc = %d, x= %d,y=%d,d= %f\n", p->iMyID, i,j,dDen);
		}
		dvx /= dDen;
		dvy /= dDen;
		la[i][j].m_dDen = dDen;
		la[i][j].m_v.m_dx = dvx;
		la[i][j].m_v.m_dy = dvy;
	}
	else
	{
		la[i][j].m_dDen = 1000;
		la[i][j].m_v.m_dx = 0;
		la[i][j].m_v.m_dy = 0;
		for (k = 0; k < 9; k++)
		{
			la[i][j].m_df[k] = 1000;
		}
	}
	if (j<CENTER + 2 && j>CENTER - 2 && i>p->iBegin_Row + 2 && j < p->iEnd_Row - 2)//����һ�������Ĥ��
	{
		if (la[i][j].m_iSol == -3 && la[i][j + 1].m_iSol != -3
			&& la[i][j - 1].m_iSol != -3 && la[i - 1][j + 1].m_iSol != -3
			&& la[i - 1][j - 1].m_iSol != -3)
		{
			la[i][j].m_dDen = (la[i][j + 1].m_dDen + la[i][j - 1].m_dDen) / 2;
			for (k = 0; k < 9; k++)
			{
				feq(&dfeq, k, la[i][j].m_dDen, 0, 0, p);
				la[i][j].m_df[k] = dfeq;
			}
			la[i][j].m_v.m_dx = 0;
			la[i][j].m_v.m_dy = 0;
		}
	}
}

__global__ void addKernelStream(lattice_str  la[WIDTH_E][HEIGHT], lattice_str  la_bak[WIDTH_E][HEIGHT], parameter* p)//�޷�����
{
	int i, j, k;
	i = blockIdx.x + p->iBegin_Row;
	j = threadIdx.x;
	if (la[i][j].m_iSol < 0 && la[i][j].m_ipr != 2)
	{
		for (k = 1; k < 9; k++)
		{
			la[i][j].m_df[k] = la_bak[i - p->iPrjx[k]][j - p->iPrjy[k]].m_df[k];
		}
	}
}

__global__ void addKernelStress(lattice_str  la[WIDTH][HEIGHT], Cuda_Segment sm1[SEGNUM], Cuda_Segment sm2[SEGNUM], parameter* p)//������
{
	int i, j;
	double v0, vj, vjj, dely;
	i = blockIdx.x;
	j = threadIdx.x;
	if (j == 0)//�±߽�
	{
		la[i][j].m_Stress = la[i][j + 1].m_v.m_dx - 0;
	}
	else if (j == HEIGHT - 1)//�ϱ߽�
	{
		la[i][j].m_Stress = 0 - la[i][HEIGHT - 2].m_v.m_dx;
	}//j=1~H-2
	else if (la[i][j + 1].m_iSol == la[i][j - 1].m_iSol || j == 1 || j == HEIGHT - 2)//ͬ��������
	{
		la[i][j].m_Stress = (la[i][j + 1].m_v.m_dx - la[i][j - 1].m_v.m_dx) / 2.0;
	}//j=1~H-2
	else if (la[i][j + 1].m_iSol*la[i][j - 1].m_iSol == 2 && la[i][j].m_iSol<0)//-1,-2
	{
		if (i >= sm1[0].m_pst.m_dx)//��ͬ��������Ĥ����Χ
		{
			if (j>HEIGHT / 2)
			{
				dely = VirticalDistance(sm1, i, j);//��Ĥ
			}
			else
			{
				dely = VirticalDistance(sm2, i, j);//��Ĥ
			}
			vj = la[i][j].m_v.m_dx;
			if (la[i][j].m_iSol == la[i][j + 1].m_iSol)
			{
				if (fabs(dely) < 0.1)
				{
					la[i][j].m_Stress = la[i][j + 1].m_v.m_dx - la[i][j].m_v.m_dx;
				}
				else
				{
					vjj = la[i][j + 1].m_v.m_dx;
					v0 = vj*(1 - dely) + dely*vjj;
					la[i][j].m_Stress = v0 / (2 * dely);
				}
			}
			else
			{
				if (fabs(dely) < 0.1)
				{
					la[i][j].m_Stress = la[i][j].m_v.m_dx - la[i][j - 1].m_v.m_dx;
				}
				else
				{
					vjj = la[i][j - 1].m_v.m_dx;
					v0 = vj*(1 - dely) + dely*vjj;
					la[i][j].m_Stress = -v0 / (2 * dely);
				}
			}
		}
	}//-1,-2,//j=1~H-2
	else if (la[i][j + 1].m_iSol >= 0 && la[i][j - 1].m_iSol < 0 && la[i][j].m_iSol < 0)//0,4
	{
		la[i][j].m_Stress = (0 - la[i][j - 1].m_v.m_dx) / 2.0;
	}
	else if (la[i][j - 1].m_iSol >= 0 && la[i][j + 1].m_iSol < 0 && la[i][j].m_iSol < 0)//0,4
	{
		la[i][j].m_Stress = (la[i][j + 1].m_v.m_dx - 0) / 2.0;
	}
	else if (la[i][j + 1].m_iSol >= 0 && la[i][j - 1].m_iSol >= 0 && la[i][j].m_iSol<0)
	{
		la[i][j].m_Stress = 0;
	}//j=1~H-2
	else if (j>HEIGHT / 2)//j=H/2~H-2
	{
		if (la[i][j - 1].m_iSol < 0)
		{
			if (la[i][j - 1].m_iSol == la[i][j - 2].m_iSol&&la[i][j - 1].m_iSol == la[i][j - 3].m_iSol)
			{
				la[i][j].m_Stress = (la[i][j - 3].m_v.m_dx - la[i][j - 1].m_v.m_dx) / 2 - la[i][j - 2].m_v.m_dx;//����
			}
			else
			{
				la[i][j].m_Stress = 0 - la[i][j - 1].m_v.m_dx;
			}
		}
		else
		{
			la[i][j].m_Stress = 0;
		}
	}//j=1~H/2
	else
	{
		if (la[i][j + 1].m_iSol < 0)
		{
			if (la[i][j + 1].m_iSol == la[i][j + 2].m_iSol&&la[i][j + 1].m_iSol == la[i][j + 3].m_iSol)
			{
				la[i][j].m_Stress = la[i][j + 2].m_v.m_dx - (la[i][j + 3].m_v.m_dx - la[i][j + 1].m_v.m_dx) / 2;//����
			}
			else
			{
				la[i][j].m_Stress = la[i][j + 1].m_v.m_dx - 0;//�Ƿ�����һ�£�
			}
		}
		else
		{
			la[i][j].m_Stress = 0;
		}
	}
}


__global__ void addKernelDeffusionNO(lattice_str  la[WIDTH_E][HEIGHT], double  NOBuf[WIDTH_E][HEIGHT], parameter* p)//�Ѿ��û������
{
	int i, j;
	double dDeltaNO, dLaplace;
	Cuda_Vector Grad, v;
	i = blockIdx.x + p->iBegin_Row;
	j = threadIdx.x;
	double Nij, N1j, Ni1, N2j, Ni2;
	Nij = la[i][j].m_dcNO;
	N1j = (i <= p->iBegin_Row&&p->iMyID == 0) ? 0 : la[i - 1][j].m_dcNO;
	Ni1 = (j > 0) ? la[i][j - 1].m_dcNO : 0;
	N2j = (i >= p->iEnd_Row - 1 && p->iMyID == p->iNumProcs - 1) ? 0 : la[i + 1][j].m_dcNO;
	Ni2 = (j<HEIGHT - 1) ? la[i][j + 1].m_dcNO : 0;
	v = la[i][j].m_v;
	if (i >= ((p->iMyID == 0) ? p->iBegin_Row + 1 : p->iBegin_Row)
		&& i <= ((p->iMyID == p->iNumProcs - 1) ? p->iEnd_Row - 2 : p->iEnd_Row - 1)
		&& j>0 && j < HEIGHT - 1)//1 - W-2, 1 - H-2
	{
		dLaplace = N1j - 2 * Nij + N2j + Ni1 - 2 * Nij + Ni2;
		if (v.m_dx>0)
		{
			Grad.m_dx = Nij - N1j;
		}
		else
		{
			Grad.m_dx = N2j - Nij;
		}
		if (v.m_dy > 0)
		{
			Grad.m_dy = Nij - Ni1;
		}
		else
		{
			Grad.m_dy = Ni2 - Nij;
		}
		dDeltaNO = p->NOSpread*dLaplace - p->NODecay*Nij - v*Grad;//dks
		NOBuf[i][j] += dDeltaNO;
	}//����
	else if (j == 0 && ((p->iMyID == 0) ? i != p->iBegin_Row : 1)
		&& ((p->iMyID == p->iNumProcs - 1) ? i != p->iEnd_Row - 1 : 1))//j=0, i!=0,i!=w-1,��ȥ��ͷ����ˮƽ��
	{
		NOBuf[i][0] += p->NOSpread*(Ni2 - 3 * Nij + N2j + N1j) - p->NODecay*Nij;
	}//��
	else if (j == HEIGHT - 1 && ((p->iMyID == 0) ? i != p->iBegin_Row : 1)
		&& ((p->iMyID == p->iNumProcs - 1) ? i != p->iEnd_Row - 1 : 1))//j=H-1,i!=0,i!=W-1,��ȥ��ͷ����ˮƽ��
	{
		NOBuf[i][HEIGHT - 1] += p->NOSpread*(Ni1 - 3 * Nij + N2j + N1j) - p->NODecay*Nij;
	}//��
	else if (i == p->iBegin_Row &&p->iMyID == 0 && j != 0 && j != HEIGHT - 1)//��ȥ��ͷ������ֱ��
	{
		NOBuf[i][j] += p->NOSpread*(N2j - 3 * Nij + Ni2 + Ni1) - p->NODecay*Nij;
	}//��
	else if (i == p->iEnd_Row - 1 && p->iMyID == p->iNumProcs - 1 && j != 0 && j != HEIGHT - 1)//��ȥ��ͷ������ֱ��
	{
		NOBuf[i][j] += p->NOSpread*(N1j - 3 * Nij + Ni2 + Ni1) - p->NODecay*Nij;
	}//��
	else if (i == p->iBegin_Row && p->iMyID == 0 && j == 0)//i=0,j=0������
	{
		NOBuf[i][j] += p->NOSpread*(Ni2 - 2 * Nij + N2j) - p->NODecay*Nij;
	}//����
	else if (i == p->iBegin_Row && p->iMyID == 0 && j == HEIGHT - 1)//i=0,j=H-1������
	{
		NOBuf[i][j] += p->NOSpread*(Ni1 - 2 * Nij + N2j) - p->NODecay*Nij;
	}//����
	else if (i == p->iEnd_Row - 1 && p->iMyID == p->iNumProcs - 1 && j == 0)//����
	{
		NOBuf[i][j] += p->NOSpread*(Ni2 - 2 * Nij + N1j) - p->NODecay*Nij;
	}//����
	else if (i == p->iEnd_Row - 1 && p->iMyID == p->iNumProcs - 1 && j == HEIGHT - 1)//����
	{
		NOBuf[i][j] += p->NOSpread*(Ni1 - 2 * Nij + N1j) - p->NODecay*Nij;
	}//����
	else
	{
		printf("NO spreading Error\n");
	}
}

__global__ void addKernelAddupNO(lattice_str  la[WIDTH_E][HEIGHT], double  NOBuf[WIDTH_E][HEIGHT], parameter* p)
{
	int i, j;
	i = blockIdx.x + p->iBegin_Row;
	j = threadIdx.x;
	double N;
	N = la[i][j].m_dcNO + NOBuf[i][j];//�ӷ������ٿ�����
	if (N<0)
	{
		//printf("NO<0,No=%f,i=%d,j=%d,type=%d,ID=%d,set to 0\n", NOBuf[i][j], i, j, la[i][j].m_bReaction,p->iMyID);
		N = 0;
	}
	/*else if (N>1)
	{
		N = 1;
	}*/
	if (i == p->iBegin_Row + 1 && p->iMyID == 0 || j == 1 || i == p->iEnd_Row - 2 && p->iMyID == p->iNumProcs - 1 || j == HEIGHT - 2)//0 �߽�
	{
		N = 0;
	}
	la[i][j].m_dcNO = N;
}

__global__ void addKernelInitMem(Cuda_Segment* mem1, Cuda_Segment* mem2, parameter * p, double * Test)
{
	int i, j;
	i = blockIdx.x;
	j = threadIdx.x;
	if (i == 0)
	{
		mem1[j].init(mem1, &j, p);
	}
	else if (i == 1)
	{
		mem2[j].init(mem2, &j, p);
	}
}

__global__ void addKernelFindLink(lattice_str  la[WIDTH_E][HEIGHT], Cuda_Segment* mem1, Cuda_Segment* mem2, lattice_link link[WIDTH_E][HEIGHT],
	int *ixn, int* iyn, int * linknum, parameter* p, double * Test)
{
	int i, j, L, m, iSol, iFol, iSeg;
	int iFlux, iFluy;
	//hipError_t cudaStatus;
	double ddy;
	int iNearest = 0;
	Cuda_Vector pp, v1, v2;
	lattice_link blink;
	i = blockIdx.x + (*ixn);
	j = threadIdx.x + (*iyn);
	link[i][j].m_i = 0;
	blink = link[i][j];
	blink.m_i = 0;
	iSol = la[i][j].m_iSol; //*Test = mem1[2].m_pst.m_dx;
	if (iSol == -1 || iSol == 0 || iSol == 4 && OnMem(i, j, mem1, mem2) || iSol == -3)
	{
		iSeg = BEGINMEM + int(double(i - p->iBegin_Row)*(p->iEnd_Mem - 1 - BEGINMEM)
			/ (p->iEnd_Row - 1 - p->iBegin_Row) + 0.5);
		for (m = 1; m < 9; m++)//m�ķ����ɹ���ָ������
		{
			iFlux = i + p->iPrjx[m];  //fluid node -1~n
			iFluy = j + p->iPrjy[m];  //iFluy -1~n
			iNearest = iSeg;
			if (iSeg < p->iEnd_Mem - 1)
			{
				if (iFluy >= (HEIGHT - 1) / 2)
				{
					ddy = fabs(mem1[iSeg + 1].m_pst.m_dy + (mem1[iSeg].m_pst.m_dy
						- mem1[iSeg + 1].m_pst.m_dy) / (mem1[iSeg].m_pst.m_dx
						- mem1[iSeg + 1].m_pst.m_dx)*(iFlux - mem1[iSeg + 1].m_pst.m_dx) - iFluy);
				}
				else
				{
					ddy = fabs(mem2[iSeg + 1].m_pst.m_dy + (mem2[iSeg].m_pst.m_dy
						- mem2[iSeg + 1].m_pst.m_dy) / (mem2[iSeg].m_pst.m_dx
						- mem2[iSeg + 1].m_pst.m_dx)*(iFlux - mem2[iSeg + 1].m_pst.m_dx) - iFluy);
				}
			}
			iFol = la[iFlux][iFluy].m_iSol;
			if (i < mem1[p->iBegin_Mem].m_pst.m_dx&&iFlux < mem1[p->iBegin_Mem].m_pst.m_dx
				|| i < mem1[p->iBegin_Mem].m_pst.m_dx&&iFlux >= mem1[p->iBegin_Mem].m_pst.m_dx&&
				iFol == -2 && iSol == -1 || i >= mem1[p->iBegin_Mem].m_pst.m_dx&&iFlux
				< mem1[p->iBegin_Mem].m_pst.m_dx&&
				iSol == -2 && iFol == -1)
			{
				continue;
			}
			if (InRegionX(iFlux, p) && iFluy >= 0 && iFluy < HEIGHT)
			{
				if (iSol == -1 && iFol == -2
					|| iSol == 0 && iFol < 0
					|| iSol == 4 && iFol < 0
					|| iSol == -3 && iFol == -2) //�ɹ���ָ�����
				{
					blink.m_ln[blink.m_i].m_iSolx = i;
					blink.m_ln[blink.m_i].m_iSoly = j;
					blink.m_ln[blink.m_i].m_iFlux = iFlux;
					blink.m_ln[blink.m_i].m_iFluy = iFluy;
					blink.m_ln[blink.m_i].m_iDir = p->iRev[m];//����Ϊ������ָ�����
					blink.m_ln[blink.m_i].m_iSegf = -1;
					for (L = (p->iMyID == 0) ? p->iBegin_Mem : p->iBegin_Mem - 1; L < ((p->iMyID == p->iNumProcs - 1) ? p->iEnd_Mem : p->iEnd_Mem + 1); L++)
					{
						if (iFluy >= (HEIGHT - 1) / 2)
						{
							v1.m_dx = iFlux; v1.m_dy = iFluy;
							v2.m_dx = i; v2.m_dy = j;
							if (_IsCross(la, &v1, &v2, &(mem1[L]), mem1[L].m_next, &pp, Test))
							{
								blink.m_ln[blink.m_i].m_bf = pp;
								blink.m_ln[blink.m_i].m_iSegf = L;
								blink.m_ln[blink.m_i].m_iType = 1;//�����ϱ�Ĥ
								if (blink.m_ln[blink.m_i].m_iDir != 3 && blink.m_ln[blink.m_i].m_iDir != 4)//����ֱ����
								{
									blink.m_ln[blink.m_i].m_dDlf = fabs(blink.m_ln[blink.m_i].m_bf.m_dx - blink.m_ln[blink.m_i].m_iFlux);
								}
								else
								{
									blink.m_ln[blink.m_i].m_dDlf = fabs(blink.m_ln[blink.m_i].m_bf.m_dy - blink.m_ln[blink.m_i].m_iFluy);
								}
								break;
							}
						}
						else
						{
							v1.m_dx = iFlux; v1.m_dy = iFluy;
							v2.m_dx = i; v2.m_dy = j;
							if (_IsCross(la, &v1, &v2, &(mem2[L]), mem2[L].m_next, &pp, Test))
							{
								blink.m_ln[blink.m_i].m_bf = pp; 
								blink.m_ln[blink.m_i].m_iSegf = L;
								blink.m_ln[blink.m_i].m_iType = -1;//�����±�Ĥ
								if (blink.m_ln[blink.m_i].m_iDir != 3 && blink.m_ln[blink.m_i].m_iDir != 4)//����ֱ����
								{
									blink.m_ln[blink.m_i].m_dDlf = fabs(blink.m_ln[blink.m_i].m_bf.m_dx - blink.m_ln[blink.m_i].m_iFlux);
								}
								else
								{
									blink.m_ln[blink.m_i].m_dDlf = fabs(blink.m_ln[blink.m_i].m_bf.m_dy - blink.m_ln[blink.m_i].m_iFluy);
								}	
								break;
							}
						}
						if (L == p->iEnd_Mem - 1 && (p->iMyID == p->iNumProcs - 1 || p->iMyID == 0))//������ڶ��϶˵㣬ʵ���޷��ж�
						{
							if (iSol >= 0)//�ڶ��϶˵�
							{
								blink.m_ln[blink.m_i].m_bf = Cuda_Vector(i, j);
								blink.m_ln[blink.m_i].m_iSegf = iNearest;
								blink.m_ln[blink.m_i].m_dDlf = 1;//�߽�պ��ڹ������;
								if (iSol == 0 && (iFol == -3 || iFol == -1))
								{
									la[iFlux][iFluy].m_bReaction = 1;
								}
							}
							else if (iSol == -1 || iSol == -3)//�ǳ�����(i,j),��������޷��ж�
							{
								blink.m_ln[blink.m_i].m_bf = Cuda_Vector(i, j);
								blink.m_ln[blink.m_i].m_iSegf = iNearest;
								blink.m_ln[blink.m_i].m_dDlf = 1;//�߽�պ��ڹ������;
								la[i][j].m_bReaction = 1;
							}
							else
							{
								printf("boundary not find Procs=%d,x=%d,y=%d,fx=%d,fy=%d,s0y=%f,s1y=%f,s2y=%f,is=%d,if=%d,%f\n", p->iMyID, i, j, iFlux, iFluy, mem1[2].m_pst.m_dy, mem2[154].m_pst.m_dy, mem2[155].m_pst.m_dy, iSol, iFol, ddy);
							}
						}
					}//for L
					if (iSol == -1 || (iSol == -3 && iFol == -2))
					{
						la[i][j].m_bReaction = 1;//��ĤҪ��NO
					}
					else if (iSol == 0 && (iFol == -3 || iFol == -1))
					{
						la[iFlux][iFluy].m_bReaction = 1;
					}
					else if ((iSol == 0 || iSol == 4) &&
						la[i - p->iPrjx[m]][j - p->iPrjy[m]].m_iSol == -1)
					{
						la[i - p->iPrjx[m]][j - p->iPrjy[m]].m_bReaction = 1;
					}
					if (blink.m_ln[blink.m_i].m_iSegf >= 0)
					{
						blink.m_i++;
					}
					if (blink.m_i > LALNNUM)
					{
						printf("Link Number> LALNNUM\n");
					}
				}//if
			}//if
		}//for m
		link[i][j] = blink;
		link[i][j].m_ior = atomicAdd(linknum, blink.m_i);
	}
}


__device__ int OnMem(int i, int j, Cuda_Segment* mem1, Cuda_Segment* mem2)
{
	if (SamePoint(i, j, mem1[LEFTVALVE].m_pst) ||
		SamePoint(i, j, mem1[RIGHTVALVE].m_pst) ||
		SamePoint(i, j, mem2[LEFTVALVE].m_pst) ||
		SamePoint(i, j, mem2[RIGHTVALVE].m_pst))
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__ int SamePoint(int i, int j, Cuda_Vector v)
{
	if (i >= v.m_dx&&i <= v.m_dx&&j >= v.m_dy&&j <= v.m_dy)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__global__ void  addKernelCopyLink(lattice_link link[WIDTH_E][HEIGHT], int *ixn, int* iyn, Cuda_Link* link0, parameter* p, double* Test)
{
	int i, j, k;
	i = blockIdx.x + *ixn;
	j = threadIdx.x + *iyn;
	for (k = 0; k < link[i][j].m_i; k++)
	{
		link0[link[i][j].m_ior + k] = link[i][j].m_ln[k];
	}
}


__global__ void addKernelReactionNO(lattice_str  la[WIDTH_E][HEIGHT], Cuda_Segment* mem1, Cuda_Segment* mem2, Valve_str * valve, double  NOBuf[WIDTH_E][HEIGHT], parameter* p)//1:Ĥ,2:�ڵ�,3:���
{
	int i, j, iSeg;
	double dks, dProduct, dstr = 0, dx;
	i = blockIdx.x + p->iBegin_Row;
	j = threadIdx.x;
	dks = p->dkNO*p->h;
	if (la[i][j].m_bReaction > 0)
	{
		if (la[i][j].m_bReaction == 1)//����Ĥ
		{
			iSeg = BEGINMEM + int(double(i - p->iBegin_Row)*(p->iEnd_Mem - 1 - BEGINMEM)
				/ (p->iEnd_Row - 1 - p->iBegin_Row) + 0.5);
			if (iSeg >= 0 && iSeg < SEGNUM_E)
			{
				if (j >= CENTER)
				{
					dstr = mem1[iSeg].m_fi;
				}
				else
				{
					dstr = mem2[iSeg].m_fi;
				}
			}
			else
			{
				printf("SegNum Wrong : %d\n", iSeg);
			}
		}
		else if (la[i][j].m_bReaction == 2)//valve��
		{
			dx = (H2 - H1) * 2.0 / VALSEGNUM;
			if (valve->m_lActive &&i < mem1[RIGHTVALVE].m_pst.m_dx)//��
			{
				iSeg = int((i - valve->m_dLx0) / dx + 0.5);
				if (iSeg >= 0 && iSeg < VALSEGNUM)
				{
					if (j >= CENTER)//��
					{
						dstr = valve->m_dULfi[iSeg];
					}
					else
					{
						dstr = valve->m_dDLfi[iSeg];
					}
				}
				else
				{
					printf("Valve left SegNum Wrong : %d\n", iSeg);
				}
			}
			else  if (valve->m_rActive)
			{
				iSeg = int((i - valve->m_dRx0) / dx + 0.5);
				if (iSeg >= 0 && iSeg < VALSEGNUM)
				{
					if (j >= CENTER)//��
					{
						dstr = valve->m_dURfi[iSeg];
					}
					else
					{
						dstr = valve->m_dDRfi[iSeg];
					}
				}
				else
				{
					printf("Valve right SegNum Wrong : %d\n", iSeg);
				}
			}
		}
		else if (la[i][j].m_bReaction == 3)//valve��
		{
			dx = (H2 - H1) * 2.0 / VALSEGNUM;
			if (valve->m_lActive &&i < mem1[RIGHTVALVE].m_pst.m_dx)//��
			{
				iSeg = int((i - valve->m_dLx0) / dx + 0.5);
				if (iSeg >= 0 && iSeg < VALSEGNUM)
				{
					if (j >= CENTER)//��
					{
						dstr = valve->m_dULfo[iSeg];
					}
					else
					{
						dstr = valve->m_dDLfo[iSeg];
					}
				}
				else
				{
					printf("Valve right SegNum Wrong : %d\n", iSeg);
				}
			}
			else if (valve->m_rActive)
			{
				iSeg = int((i - valve->m_dRx0) / dx + 0.5);
				if (iSeg >= 0 && iSeg < VALSEGNUM)
				{
					if (j >= CENTER)//��
					{
						dstr = valve->m_dURfo[iSeg];
					}
					else
					{
						dstr = valve->m_dDRfo[iSeg];
					}
				}
				else
				{
					printf("Valve right SegNum Wrong : %d\n", iSeg);
				}
			}
		}
		else
		{
			dstr = 0;
		}
		dProduct = dks*fabs(dstr) / (la[i][j].m_dDen*(2 * p->Tau - 1) / 6);
		/*if (!(la[i][j].m_dcNO <= LIMITNO) && dProduct > p->NODecay)
		{
			dProduct = p->NODecay;
		}*/
		NOBuf[i][j] = dProduct;
		la[i][j].m_bReaction = 0;
	}
	else
	{
		NOBuf[i][j] = 0;
	}
}



__device__ int  _IsCross(lattice_str  la[WIDTH_E][HEIGHT], Cuda_Vector* l1, Cuda_Vector* l2, Cuda_Segment* seg, Cuda_Segment* next, Cuda_Vector *pp, double *Test)//�ж�s1s2��l1l2�Ľ���
{
	Cuda_Vector s1, s2;
	if (next != NULL)
	{
		double f1, f2, f3, f4;
		double a1, b1, c1, a2, b2, c2;
		s1 = seg->m_pst;
		s2 = next->m_pst;
		a1 = s2.m_dy - s1.m_dy;
		b1 = -(s2.m_dx - s1.m_dx);
		c1 = -a1*s1.m_dx - b1*s1.m_dy;
		a2 = l2->m_dy - l1->m_dy;
		b2 = -(l2->m_dx - l1->m_dx);
		c2 = -a2*l1->m_dx - b2*l1->m_dy;
		f1 = (*l1 - s1) ^ (s2 - s1);//s1,l1,s2
		f2 = (*l2 - s1) ^ (s2 - s1);//s1,l2,s2
		f3 = (s2 - *l1) ^ (*l2 - *l1);//l1,s2,l2
		f4 = (s1 - *l1) ^ (*l2 - *l1);//l1,s1,l2
		if (f1*f2 < 0 && f3*f4<0 || f1 == 0 && _OnSegment(&s1, &s2, l1) || f2 == 0 && _OnSegment(&s1, &s2, l2)
			|| f3 == 0 && _OnSegment(l1, l2, &s2) || f4 == 0 && _OnSegment(l1, l2, &s1))//s2�ϵĲ�Ҫ
		{
			if (pp != NULL)
			{
				if (a1*b2 - a2*b1>0 || a1*b2 - a2*b1 < 0)
				{
					pp->m_dx = (c2*b1 - c1*b2) / (a1*b2 - a2*b1);
					pp->m_dy = (c1*a2 - c2*a1) / (a1*b2 - a2*b1);
					pp->m_dx = _ErrorCorrect(pp->m_dx, l1->m_dx, l2->m_dx);
					pp->m_dy = _ErrorCorrect(pp->m_dy, l1->m_dy, l2->m_dy);
				}
				else
				{
					if (la[int(l1->m_dx)][int(l1->m_dy)].m_iSol == -1)//�����ص���ֱ����Ϊ�����ڹ�����ϣ�����
					{
						*pp = *l1;
					}
					else
					{
						*pp = *l2;
					}
				}
			}
			return 1;
		}
		else
		{
			return 0;
		}
	}
	else
	{
		return 0;
	}
}

__device__ int  _OnSegment(Cuda_Vector* p1, Cuda_Vector* p2, Cuda_Vector* p3)
{
	double dxm, dxn, dym, dyn;
	if (p1->m_dx < p2->m_dx)
	{
		dxn = p1->m_dx;
		dxm = p2->m_dx;
	}
	else
	{
		dxn = p2->m_dx;
		dxm = p1->m_dx;
	}
	if (p1->m_dy < p2->m_dy)
	{
		dyn = p1->m_dy;
		dym = p2->m_dy;
	}
	else
	{
		dyn = p2->m_dy;
		dym = p1->m_dy;
	}
	if (p3->m_dx<dxn || p3->m_dx>dxm || p3->m_dy<dyn || p3->m_dy>dym)
	{
		return 0;
	}
	else
	{
		return 1;
	}
}

__device__ double _ErrorCorrect(double x, double x1, double x2)//��֪xλ��x1��x2֮�䣬����x������������Χ���ܵ��������������������
{
	if (x1 <= x2)
	{
		if (x <= x1)
		{
			return  x1;
		}
		else if (x >= x2)
		{
			return x2;
		}
		else
		{
			return x;
		}
	}
	else
	{
		if (x <= x2)
		{
			return x2;
		}
		else if (x >= x1)
		{
			return x1;
		}
		else
		{
			return x;
		}
	}
}


__device__ double VirticalDistance(Cuda_Segment sm[SEGNUM_E], int i, int j)
{
	int m, ix, ii;
	double dely;
	ix = int((i - sm[0].m_pst.m_dx)*SEGNUM_E / (sm[SEGNUM_E - 1].m_pst.m_dx - sm[0].m_pst.m_dx));
	ii = (ix - 1 > 0) ? ix - 1 : 0;
	if (ix >= SEGNUM_E - 1) ix = SEGNUM_E - 1;
	for (m = ii; m < ix + 1; m++)
	{
		if (m < SEGNUM_E - 1)
		{
			if (i >= sm[m].m_pst.m_dx && i <= sm[m + 1].m_pst.m_dx)
			{
				break;
			}
		}
	}
	dely = sm[m].m_pst.m_dy + (sm[m + 1].m_pst.m_dy - sm[m].m_pst.m_dy) /
		(sm[m + 1].m_pst.m_dx - sm[m].m_pst.m_dx)*(i - sm[m].m_pst.m_dx) - j;
	return fabs(dely);
}

__device__ int InRegionX(double x, parameter* p)
{
	if (p->iMyID > 0 && p->iMyID < p->iNumProcs - 1 && x >= 0 && x < WIDTH_E)
	{
		return 1;
	}
	else if (p->iMyID == 0 && x >= p->iBegin_Row &&  x < WIDTH_E)
	{
		return 1;
	}
	else if (p->iMyID == p->iNumProcs - 1 && x >= 0 && x < p->iEnd_Row)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

extern "C" double GetValue()
{
	hipMemcpy(&dTest, dev_test, sizeof(double), hipMemcpyDeviceToHost);
	return double(dTest);

}

extern "C"  void freeMem()
{
	hipFree(dev_p);
	//hipFree(dev_a);
	hipFree(dev_a_bak);
	//hipFree(dev_NOBuf);
	hipFree(dev_sm1);
	hipFree(dev_sm2);
	//hipFree(dev_link);
	hipFree(dev_linknum);
	hipFree(dev_laln);
	hipFree(dev_ixn);
	hipFree(dev_iyn);
	hipFree(dev_valve);
	hipFree(dev_test);
}


/*void Show1(int Par, char*ch)
{
char cc[50];
sprintf(cc, "%d", Par);
MessageBox(NULL, ch, cc, 1);
}

void Show1(double Par, char*ch)
{
char cc[50];
sprintf(cc, "%16.15e", Par);
//	MessageBox(NULL, ch, cc, MB_OK);
}

void Show1(float Par, char*ch)
{
char cc[50];
sprintf(cc, "%f", Par);
//	MessageBox(NULL, ch, cc, MB_OK);
}*/
